#include "hip/hip_runtime.h"
#include "CudaMatrixGPU.h"
#include "CudaMatrixGPUKernel.cu"

#include <cassert>

#define MAX_GRID_DIMENSION (65535)


//parallelization is used only on the rows.
//this function assumes that m is large, otherwise, you get poor utilization of the gpu.
//in order to optimize this function for smaller matrix heights, a different approach should be taken and parallelism should be exploited along the columns as well.
//this can be done by using reduction
bool cuMultMatrixByVectorComplex(int m, int n, int numPitchedRows, const std::complex<float>* d_A, const std::complex<float>* d_X, std::complex<float>* d_b)
{
	if(m <= 0 || n <= 0)
	{
		return false;
	}

	hipMemset(d_b, 0, sizeof(float2)*m);

	const unsigned int blockSize = 256;

	int numBlocks = m / blockSize + 1;
	int numCycles = n / blockSize + 1;

	dim3 threads(blockSize, 1);
	dim3 grid(numBlocks, 1);
	multMatrixByVectorComplexKernel<blockSize><<<grid, threads>>>(m, n, numPitchedRows, numCycles, (const float2*)d_A, (const float2*)d_X, (float2*)d_b);

	return true;
}

bool cuMultMatrixByVectorReal(int m, int n, int numPitchedRows, const float* d_A, const float* d_X, float* d_b)
{
	if(m <= 0 || n <= 0)
	{
		return false;
	}

	hipMemset(d_b, 0, sizeof(float)*m);

	const unsigned int blockSize = 256;

	int numBlocks = m / blockSize + 1;
	int numCycles = n / blockSize + 1;

	dim3 threads(blockSize, 1);
	dim3 grid(numBlocks, 1);
	multMatrixByVectorRealKernel<blockSize><<<grid, threads>>>(m, n, numPitchedRows, numCycles, d_A, d_X, d_b);

	return true;
}

bool cuAddVectorsComplex(int numElements, const std::complex<float>* d_A, const std::complex<float>* d_B, std::complex<float>* d_C)
{
	if(numElements <= 0)
	{
		return false;
	}
	
	const unsigned int blockSize = 256;
	int numBlocks = I_DIV_UP(numElements, blockSize);
	assert(numBlocks > 0 && numBlocks <= MAX_GRID_DIMENSION);

	dim3 threads(blockSize, 1);
	dim3 grid(numBlocks, 1);
	addVectorsComplexKernel<blockSize><<<grid, threads>>>(numElements, (const float2*)d_A, (const float2*)d_B, (float2*)d_C);

	return true;
}

bool cuAddVectorsReal(int numElements, const float* d_A, const float* d_B, float* d_C)
{
	if(numElements <= 0)
	{
		return false;
	}

	const unsigned int blockSize = 256;
	int numBlocks = I_DIV_UP(numElements, blockSize);
	assert(numBlocks > 0 && numBlocks <= MAX_GRID_DIMENSION);

	dim3 threads(blockSize, 1);
	dim3 grid(numBlocks, 1);
	addVectorsRealKernel<blockSize><<<grid, threads>>>(numElements, d_A, d_B, d_C);

	return true;
}

bool cuSubVectorsComplex(int numElements, const std::complex<float>* d_A, const std::complex<float>* d_B, std::complex<float>* d_C)
{
	if(numElements <= 0)
	{
		return false;
	}

	const unsigned int blockSize = 256;
	int numBlocks = I_DIV_UP(numElements, blockSize);
	assert(numBlocks > 0 && numBlocks <= MAX_GRID_DIMENSION);

	dim3 threads(blockSize, 1);
	dim3 grid(numBlocks, 1);
	subVectorsComplexKernel<blockSize><<<grid, threads>>>(numElements, (const float2*)d_A, (const float2*)d_B, (float2*)d_C);

	return true;
}

bool cuSubVectorsComplexDouble(int numElements, const std::complex<double>* d_A, const std::complex<double>* d_B, std::complex<double>* d_C)
{
	if (numElements <= 0)
	{
		return false;
	}

	const unsigned int blockSize = 256;
	int numBlocks = I_DIV_UP(numElements, blockSize);
	assert(numBlocks > 0 && numBlocks <= MAX_GRID_DIMENSION);

	dim3 threads(blockSize, 1);
	dim3 grid(numBlocks, 1);
	subVectorsComplexDoubleKernel<blockSize> << <grid, threads >> >(numElements, (const double2*)d_A, (const double2*)d_B, (double2*)d_C);

	return true;
}
bool cuSubVectorsReal(int numElements, const float* d_A, const float* d_B, float* d_C)
{
	if(numElements <= 0)
	{
		return false;
	}

	const unsigned int blockSize = 256;

	const int maxNumElements = blockSize*MAX_GRID_DIMENSION; //the maximum grid size in each dimension is 65535 so if we want to use 1D grid we should take this into account

	int numCycles = I_DIV_UP(numElements, maxNumElements);
	dim3 threads(blockSize, 1);

	for(int i = 0; i < numCycles - 1; i++)
	{
		int numBlocksInCurrentCycle = MAX_GRID_DIMENSION;
		dim3 grid(numBlocksInCurrentCycle, 1);
		subVectorsRealKernel<blockSize><<<grid, threads>>>(maxNumElements, d_A, d_B, d_C);
		d_A +=  (i + 1)*maxNumElements;
		d_B +=  (i + 1)*maxNumElements;
		d_C +=  (i + 1)*maxNumElements;
	}
	int numElementsLeft = numElements - (numCycles - 1)*maxNumElements;
	int numBlocksLeft = I_DIV_UP(numElementsLeft, blockSize);
	dim3 grid(numBlocksLeft, 1);
	subVectorsRealKernel<blockSize><<<grid, threads>>>(numElementsLeft, d_A, d_B, d_C);

	return true;
}

bool cuScaleVectorComplex(int numElements, std::complex<float>* d_A, std::complex<float> alpha)
{
	if(numElements <= 0)
	{
		return false;
	}

	const unsigned int blockSize = 256;
	int numBlocks = I_DIV_UP(numElements, blockSize);
	assert(numBlocks > 0 && numBlocks <= MAX_GRID_DIMENSION);

	dim3 threads(blockSize, 1);
	dim3 grid(numBlocks, 1);
	float2 a;
	a.x = alpha.real();
	a.y = alpha.imag();

	scaleVectorComplexKernel<blockSize><<<grid, threads>>>(numElements, (float2*)d_A, a);

	return true;
}

bool cuScaleVectorComplexDouble(int numElements, std::complex<double>* d_A, std::complex<double> alpha)
{
	if (numElements <= 0)
	{
		return false;
	}

	const unsigned int blockSize = 256;
	int numBlocks = I_DIV_UP(numElements, blockSize);
	assert(numBlocks > 0 && numBlocks <= MAX_GRID_DIMENSION);

	dim3 threads(blockSize, 1);
	dim3 grid(numBlocks, 1);
	double2 a;
	a.x = alpha.real();
	a.y = alpha.imag();

	scaleVectorComplexDoubleKernel<blockSize> << <grid, threads >> >(numElements, (double2*)d_A, a);

	return true;
}
bool cuExponentVectorComplex(int numElements, std::complex<float>* d_A)
{
	if(numElements <= 0)
	{
		return false;
	}

	const unsigned int blockSize = 256;
	int numBlocks = I_DIV_UP(numElements, blockSize);
	assert(numBlocks > 0 && numBlocks <= MAX_GRID_DIMENSION);

	dim3 threads(blockSize, 1);
	dim3 grid(numBlocks, 1);

	exponentVectorComplexKernel<blockSize><<<grid, threads>>>(numElements, (float2*)d_A);

	return true;
}

bool cuExponentVectorComplexDouble(int numElements, std::complex<double>* d_A)
{
	if (numElements <= 0)
	{
		return false;
	}

	const unsigned int blockSize = 256;
	int numBlocks = I_DIV_UP(numElements, blockSize);
	assert(numBlocks > 0 && numBlocks <= MAX_GRID_DIMENSION);

	dim3 threads(blockSize, 1);
	dim3 grid(numBlocks, 1);

	exponentVectorComplexDoubleKernel<blockSize> << <grid, threads >> >(numElements, (double2*)d_A);

	return true;
}

bool cuConvertRealToComplex(int numElements, std::complex<float>* d_A, const float* d_B)
{
	if(numElements <= 0)
	{
		return false;
	}

	const unsigned int blockSize = 256;
	int numBlocks = I_DIV_UP(numElements, blockSize);
	assert(numBlocks > 0 && numBlocks <= MAX_GRID_DIMENSION);

	dim3 threads(blockSize, 1);
	dim3 grid(numBlocks, 1);

	convertRealToComplexKernel<blockSize><<<grid, threads>>>(numElements, (float2*)d_A, d_B);

	return true;
}

bool cuConjugate(int numElements, std::complex<float>* d_A)
{
	if(numElements <= 0)
	{
		return false;
	}

	const unsigned int blockSize = 256;
	int numBlocks = I_DIV_UP(numElements, blockSize);
	assert(numBlocks > 0 && numBlocks <= MAX_GRID_DIMENSION);

	dim3 threads(blockSize, 1);
	dim3 grid(numBlocks, 1);

	conjugateComplexKernel<blockSize><<<grid, threads>>>(numElements, (float2*)d_A);

	return true;
}

bool cuAddComplexScalar(int numElements, std::complex<float>* d_A, std::complex<float> alpha)
{
	if(numElements <= 0)
	{
		return false;
	}

	const unsigned int blockSize = 256;

	const int maxNumElements = blockSize*MAX_GRID_DIMENSION; //the maximum grid size in each dimension is 65535 so if we want to use 1D grid we should take this into account

	int numCycles = I_DIV_UP(numElements, maxNumElements);
	dim3 threads(blockSize, 1);

	float2 a;
	a.x = alpha.real();
	a.y = alpha.imag();

	for(int i = 0; i < numCycles - 1; i++)
	{
		int numBlocksInCurrentCycle = MAX_GRID_DIMENSION;
		dim3 grid(numBlocksInCurrentCycle, 1);
		addComplexScalarKernel<blockSize><<<grid, threads>>>(maxNumElements, (float2*)d_A, a);
		d_A +=  (i + 1)*maxNumElements;
	}
	int numElementsLeft = numElements - (numCycles - 1)*maxNumElements;
	int numBlocksLeft = I_DIV_UP(numElementsLeft, blockSize);
	dim3 grid(numBlocksLeft, 1);
	addComplexScalarKernel<blockSize><<<grid, threads>>>(numElementsLeft, (float2*)d_A, a);

	return true;
}

bool cuAddRealScalar(int numElements, float* d_A, float alpha)
{
	if(numElements <= 0)
	{
		return false;
	}

	const unsigned int blockSize = 256;
	int numBlocks = I_DIV_UP(numElements, blockSize);
	assert(numBlocks > 0 && numBlocks <= MAX_GRID_DIMENSION);

	dim3 threads(blockSize, 1);
	dim3 grid(numBlocks, 1);

	addRealScalarKernel<blockSize><<<grid, threads>>>(numElements, d_A, alpha);

	return true;
}

