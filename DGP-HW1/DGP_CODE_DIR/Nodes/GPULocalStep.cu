#include "hip/hip_runtime.h"
#include "GPULocalStep.h"
#include "GPULocalStepKernel.cu"

#include <cassert>

#define MAX_GRID_DIMENSION (65535)

bool cuProjectPointsToPolygonNoK(int numElements, std::complex<double>* log_fz, std::complex<double>* nu_f, const double log_SigmaA, const double sigmaB, const double k, const double xIntersection, const double epsilon, const double m){
	if (numElements <= 0)
	{
		return false;
	}

	const unsigned int blockSize = 256;
	int numBlocks = I_DIV_UP(numElements, blockSize);
	assert(numBlocks > 0 && numBlocks <= MAX_GRID_DIMENSION);

	dim3 threads(blockSize, 1);
	dim3 grid(numBlocks, 1);
	projectPointsToPolygonNokKernel << <grid, threads >> >(numElements, (double2*)log_fz, (double2*)nu_f, log_SigmaA, sigmaB, k, xIntersection, epsilon, m);

	return true;

}

bool cuProjectPointsToPolygonNoK_HP(int numElements, std::complex<double>* x_vec, const double log_SigmaA, const double sigmaB, const double k, const double xIntersection, const double epsilon, const double m){
	if (numElements <= 0)
	{
		return false;
	}

	const unsigned int blockSize = 256;
	int numBlocks = I_DIV_UP(numElements, blockSize);
	assert(numBlocks > 0 && numBlocks <= MAX_GRID_DIMENSION);

	dim3 threads(blockSize, 1);
	dim3 grid(numBlocks, 1);
	projectPointsToPolygonNokKernel_HP << <grid, threads >> >(numElements, (double2*)x_vec, log_SigmaA, sigmaB, k, xIntersection, epsilon, m);

	return true;

}

bool cuProjectPointsToPolygonWithK(int numElements, std::complex<double>* log_fz, std::complex<double>* nu_f, const double log_SigmaA, const double sigmaB, const double k, const double epsilon, const double m){
	if (numElements <= 0)
	{
		return false;
	}

	const unsigned int blockSize = 256;
	int numBlocks = I_DIV_UP(numElements, blockSize);
	assert(numBlocks > 0 && numBlocks <= MAX_GRID_DIMENSION);

	dim3 threads(blockSize, 1);
	dim3 grid(numBlocks, 1);
	projectPointsToPolygonWithkKernel << <grid, threads >> >(numElements, (double2*)log_fz, (double2*)nu_f, log_SigmaA, sigmaB, k, epsilon, m);

	return true;
}

bool cuProjectPointsToPolygonWithK_HP(int numElements, std::complex<double>* x_vec, const double log_SigmaA, const double sigmaB, const double k, const double epsilon, const double m){
	if (numElements <= 0)
	{
		return false;
	}

	const unsigned int blockSize = 256;
	int numBlocks = I_DIV_UP(numElements, blockSize);
	assert(numBlocks > 0 && numBlocks <= MAX_GRID_DIMENSION);

	dim3 threads(blockSize, 1);
	dim3 grid(numBlocks, 1);
	projectPointsToPolygonWithkKernel_HP << <grid, threads >> >(numElements, (double2*)x_vec, log_SigmaA, sigmaB, k, epsilon, m);

	return true;
}

bool cuProjectPointToPolygonMinSeg(int numElements, std::complex<double>* log_fz, std::complex<double>* nu_f, const double* mXvaluesOfIntersections, const double* mYvaluesOfIntersections, const int NumOfsegments, const int epsilon){

	if (numElements <= 0)
	{
		return false;
	}

	const unsigned int blockSize = 256;
	int numBlocks = I_DIV_UP(numElements, blockSize);
	assert(numBlocks > 0 && numBlocks <= MAX_GRID_DIMENSION);

	dim3 threads(blockSize, 1);
	dim3 grid(numBlocks, 1);
	projectPointToPolygonMinSegKernel <<<grid, threads >>>(numElements, (double2*)log_fz, (double2*)nu_f, mXvaluesOfIntersections, mYvaluesOfIntersections, NumOfsegments, epsilon);

	return true;

}

bool cuProjectPointToPolygonMinSeg_HP(int numElements, std::complex<double>* x_vec, const double* mXvaluesOfIntersections, const double* mYvaluesOfIntersections, const int NumOfsegments, const int epsilon){

	if (numElements <= 0)
	{
		return false;
	}

	const unsigned int blockSize = 256;
	int numBlocks = I_DIV_UP(numElements, blockSize);
	assert(numBlocks > 0 && numBlocks <= MAX_GRID_DIMENSION);

	dim3 threads(blockSize, 1);
	dim3 grid(numBlocks, 1);
	projectPointToPolygonMinSegKernel_HP << <grid, threads >> >(numElements, (double2*)x_vec, mXvaluesOfIntersections, mYvaluesOfIntersections, NumOfsegments, epsilon);

	return true;

}
