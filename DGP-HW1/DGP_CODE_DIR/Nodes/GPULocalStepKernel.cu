
#include <hip/hip_runtime.h>
#include "math.h"

#ifndef _GPU_LOCAL_STEP_KERNEL_
#define _GPU_LOCAL_STEP_KERNEL_

#define MAX_NUMBER_OF_SEGMENTS 3

/*__shared__ bool allPointsInPolygon = true;
__global__ void CheckStopCondition(const int numElements, double2* log_fz, double2* nu_f, const double log_SigmaA, const double sigmaB, const double k, const double epsilon, const double m){
//use reduction?
}*/
__global__ void projectPointsToPolygonNokKernel(const int numElements, double2* log_fz, double2* nu_f, const double log_SigmaA, const double sigmaB, const double k, const double xIntersection, const double epsilon, const double m)
{
	const int index = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (index < numElements)
	{
		double2 log_fz_cur = log_fz[index];
		double2 nu_f_cur = nu_f[index];
		double x = sqrt(pow(nu_f_cur.x, 2) + pow(nu_f_cur.y, 2));
		double y = log_fz_cur.x;
		const double prevX = x;
		const double Yintersection = -log((1 - xIntersection) / sigmaB);

		if (!((x <= k + epsilon) && (x <= log_SigmaA - y + epsilon) && (log(sigmaB) + m*x <= y + epsilon))){
			if (y >= x + log_SigmaA){
				x = 0;
				y = log_SigmaA;
			}
			else if (y > x + Yintersection - xIntersection){
				x = (x - y + log_SigmaA) / 2;
				y = (y - prevX + log_SigmaA) / 2;
			}
			else if (y >= -x / m + Yintersection + xIntersection / m){
				x = xIntersection;
				y = Yintersection;
			}
			else if (y > -x / m + log(sigmaB)){
				x = (x + m*y - m*log(sigmaB)) / (pow(m, 2) + 1);
				y = m*((prevX + m*y - m*log(sigmaB)) / (pow(m, 2) + 1)) + log(sigmaB);
			}
			else{
				x = 0;
				y = log(sigmaB);
			}
			nu_f_cur.x = (x / prevX)*(nu_f_cur.x);
			nu_f_cur.y = (x / prevX)*(nu_f_cur.y);
			nu_f[index] = nu_f_cur;

			log_fz_cur.x = y;
			log_fz_cur.y = log_fz_cur.y;
			log_fz[index] = log_fz_cur;
		}
		//else - update if point is in polygon somehow
	}

}

__global__ void projectPointsToPolygonNokKernel_unsplit(const int numElements, double2* x_vec, const double log_SigmaA, const double sigmaB, const double k, const double xIntersection, const double epsilon, const double m)
{
	const int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index < numElements)
	{
		double2 log_fz_cur = x_vec[index];
		double2 nu_f_cur = x_vec[index + numElements];
		double x = sqrt(pow(nu_f_cur.x, 2) + pow(nu_f_cur.y, 2));
		double y = log_fz_cur.x;
		const double prevX = x;
		const double Yintersection = -log((1 - xIntersection) / sigmaB);

		if (!((x <= k + epsilon) && (x <= log_SigmaA - y + epsilon) && (log(sigmaB) + m*x <= y + epsilon))){
			if (y >= x + log_SigmaA){
				x = 0;
				y = log_SigmaA;
			}
			else if (y > x + Yintersection - xIntersection){
				x = (x - y + log_SigmaA) / 2;
				y = (y - prevX + log_SigmaA) / 2;
			}
			else if (y >= -x / m + Yintersection + xIntersection / m){
				x = xIntersection;
				y = Yintersection;
			}
			else if (y > -x / m + log(sigmaB)){
				x = (x + m*y - m*log(sigmaB)) / (pow(m, 2) + 1);
				y = m*((prevX + m*y - m*log(sigmaB)) / (pow(m, 2) + 1)) + log(sigmaB);
			}
			else{
				x = 0;
				y = log(sigmaB);
			}
			nu_f_cur.x = (x / prevX)*(nu_f_cur.x);
			nu_f_cur.y = (x / prevX)*(nu_f_cur.y);
			x_vec[index + numElements] = nu_f_cur;

			log_fz_cur.x = y;
			log_fz_cur.y = log_fz_cur.y;
			x_vec[index] = log_fz_cur;
		}
		//else - update if point is in polygon somehow
	}

}

__global__ void projectPointsToPolygonWithkKernel(const int numElements, double2* log_fz, double2* nu_f, const double log_SigmaA, const double sigmaB, const double k, const double epsilon, const double m)
{
	const int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index < numElements)
	{
		double2 log_fz_cur = log_fz[index];
		double2 nu_f_cur = nu_f[index];

		double x = sqrt(pow(nu_f_cur.x, 2) + pow(nu_f_cur.y, 2));
		double y = log_fz_cur.x;
		const double prevX = x;

		if (!((x <= k + epsilon) && (x <= log_SigmaA - y + epsilon) && (log(sigmaB) + m*x <= y + epsilon))){
			if (y >= x + log_SigmaA){
				x = 0;
				y = log_SigmaA;
			}
			else if (y > x + log_SigmaA - 2 * k){
				x = (x - y + log_SigmaA) / 2;
				y = (y - prevX + log_SigmaA) / 2;
			}
			else if (y >= log_SigmaA - k){
				x = k;
				y = log_SigmaA;
			}
			else if (y > log(sigmaB / (1 - k))){
				x = k;
			}
			else if (y >= -x / m + log(sigmaB / (1 - k)) + k / m){
				x = k;
				y = log(sigmaB / (1 - k));
			}
			else if (y > -x / m + log(sigmaB)){
				double prevX = x;
				x = (x + m*y - m*log(sigmaB)) / (pow(m, 2) + 1);
				y = m*((prevX + m*y - m*log(sigmaB)) / (pow(m, 2) + 1)) + log(sigmaB);
			}
			else{
				x = 0;
				y = log(sigmaB);
			}
			nu_f_cur.x = (x / prevX)*(nu_f_cur.x);
			nu_f_cur.y = (x / prevX)*(nu_f_cur.y);
			nu_f[index] = nu_f_cur;

			log_fz_cur.x = y;
			log_fz_cur.y = log_fz_cur.y;
			log_fz[index] = log_fz_cur;
		}
		//else - update if point is in polygon somehow
	}
}

__global__ void projectPointsToPolygonWithkKernel_unsplit(const int numElements, double2* x_vec, const double log_SigmaA, const double sigmaB, const double k, const double epsilon, const double m)
{
	const int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index < numElements)
	{
		double2 log_fz_cur = x_vec[index];
		double2 nu_f_cur = x_vec[index + numElements];

		double x = sqrt(pow(nu_f_cur.x, 2) + pow(nu_f_cur.y, 2));
		double y = log_fz_cur.x;
		const double prevX = x;

		if (!((x <= k + epsilon) && (x <= log_SigmaA - y + epsilon) && (log(sigmaB) + m*x <= y + epsilon))){
			if (y >= x + log_SigmaA){
				x = 0;
				y = log_SigmaA;
			}
			else if (y > x + log_SigmaA - 2 * k){
				x = (x - y + log_SigmaA) / 2;
				y = (y - prevX + log_SigmaA) / 2;
			}
			else if (y >= log_SigmaA - k){
				x = k;
				y = log_SigmaA;
			}
			else if (y > log(sigmaB / (1 - k))){
				x = k;
			}
			else if (y >= -x / m + log(sigmaB / (1 - k)) + k / m){
				x = k;
				y = log(sigmaB / (1 - k));
			}
			else if (y > -x / m + log(sigmaB)){
				double prevX = x;
				x = (x + m*y - m*log(sigmaB)) / (pow(m, 2) + 1);
				y = m*((prevX + m*y - m*log(sigmaB)) / (pow(m, 2) + 1)) + log(sigmaB);
			}
			else{
				x = 0;
				y = log(sigmaB);
			}
			nu_f_cur.x = (x / prevX)*(nu_f_cur.x);
			nu_f_cur.y = (x / prevX)*(nu_f_cur.y);
			x_vec[index + numElements] = nu_f_cur;

			log_fz_cur.x = y;
			log_fz_cur.y = log_fz_cur.y;
			x_vec[index] = log_fz_cur;
		}
		//else - update if point is in polygon somehow
	}
}

__global__ void projectPointToPolygonMinSegKernel(const int numElements, double2* log_fz, double2* nu_f, const double* mXvaluesOfIntersections, const double* mYvaluesOfIntersections, const int NumOfsegments, const int epsilon){

	const int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index < numElements)
	{
		double2 log_fz_cur = log_fz[index];
		double2 nu_f_cur = nu_f[index];

		double x = sqrt(pow(nu_f_cur.x, 2) + pow(nu_f_cur.y, 2));
		double y = log_fz_cur.x;
		const double prevX = x;
		bool isInsidePolygom = true;
		int i;

		for (i = 0; i < NumOfsegments; i++){
			double crossSegValue = (mXvaluesOfIntersections[i + 1] - mXvaluesOfIntersections[i])*(y - mYvaluesOfIntersections[i]) - (mYvaluesOfIntersections[i + 1] - mYvaluesOfIntersections[i])*(x - mXvaluesOfIntersections[i]);
			if (crossSegValue > 0 - epsilon){
				isInsidePolygom = false;
			}
		}

		if (!isInsidePolygom){

			double closestPointsXvalues[MAX_NUMBER_OF_SEGMENTS];
			double closestPointsYvalues[MAX_NUMBER_OF_SEGMENTS];
			double minDistances[MAX_NUMBER_OF_SEGMENTS];

			//using cross product
			for (i = 0; i < NumOfsegments; i++){
				double dot = (x - mXvaluesOfIntersections[i])*(mXvaluesOfIntersections[i + 1] - mXvaluesOfIntersections[i]) + (y - mYvaluesOfIntersections[i])*(mYvaluesOfIntersections[i + 1] - mYvaluesOfIntersections[i]);
				double projectionOnLine = dot / (pow((mXvaluesOfIntersections[i + 1] - mXvaluesOfIntersections[i]), 2) + pow((mYvaluesOfIntersections[i + 1] - mYvaluesOfIntersections[i]), 2));
				double t = fmax(0.0, fmin(1.0, projectionOnLine));
				double closestPointXvalue = mXvaluesOfIntersections[i] + t*(mXvaluesOfIntersections[i + 1] - mXvaluesOfIntersections[i]);
				double closestPointYvalue = mYvaluesOfIntersections[i] + t*(mYvaluesOfIntersections[i + 1] - mYvaluesOfIntersections[i]);
				closestPointsXvalues[i] = closestPointXvalue;
				closestPointsYvalues[i] = closestPointYvalue;
				minDistances[i] = pow((x - closestPointXvalue), 2) + pow((y - closestPointYvalue), 2);
			}
			int indexOfMin = 0;
			for (i = 1; i < NumOfsegments; i++)
				indexOfMin = (minDistances[i] < minDistances[indexOfMin] ? i : indexOfMin);

			x = closestPointsXvalues[indexOfMin];
			y = closestPointsYvalues[indexOfMin];

			nu_f_cur.x = (x / prevX)*(nu_f_cur.x);
			nu_f_cur.y = (x / prevX)*(nu_f_cur.y);
			nu_f[index] = nu_f_cur;

			log_fz_cur.x = y;
			log_fz_cur.y = log_fz_cur.y;
			log_fz[index] = log_fz_cur;

		}//else - update if point is in polygon somehow
	}
}

__global__ void projectPointToPolygonMinSegKernel_unsplit(const int numElements, double2* x_vec, const double* mXvaluesOfIntersections, const double* mYvaluesOfIntersections, const int NumOfsegments, const int epsilon){

	const int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index < numElements)
	{
		double2 log_fz_cur = x_vec[index];
		double2 nu_f_cur = x_vec[index + numElements];

		double x = sqrt(pow(nu_f_cur.x, 2) + pow(nu_f_cur.y, 2));
		double y = log_fz_cur.x;
		const double prevX = x;
		bool isInsidePolygom = true;
		int i;

		for (i = 0; i < NumOfsegments; i++){
			double crossSegValue = (mXvaluesOfIntersections[i + 1] - mXvaluesOfIntersections[i])*(y - mYvaluesOfIntersections[i]) - (mYvaluesOfIntersections[i + 1] - mYvaluesOfIntersections[i])*(x - mXvaluesOfIntersections[i]);
			if (crossSegValue > 0 - epsilon){
				isInsidePolygom = false;
			}
		}

		if (!isInsidePolygom){

			double closestPointsXvalues[MAX_NUMBER_OF_SEGMENTS];
			double closestPointsYvalues[MAX_NUMBER_OF_SEGMENTS];
			double minDistances[MAX_NUMBER_OF_SEGMENTS];

			//using cross product
			for (i = 0; i < NumOfsegments; i++){
				double dot = (x - mXvaluesOfIntersections[i])*(mXvaluesOfIntersections[i + 1] - mXvaluesOfIntersections[i]) + (y - mYvaluesOfIntersections[i])*(mYvaluesOfIntersections[i + 1] - mYvaluesOfIntersections[i]);
				double projectionOnLine = dot / (pow((mXvaluesOfIntersections[i + 1] - mXvaluesOfIntersections[i]), 2) + pow((mYvaluesOfIntersections[i + 1] - mYvaluesOfIntersections[i]), 2));
				double t = fmax(0.0, fmin(1.0, projectionOnLine));
				double closestPointXvalue = mXvaluesOfIntersections[i] + t*(mXvaluesOfIntersections[i + 1] - mXvaluesOfIntersections[i]);
				double closestPointYvalue = mYvaluesOfIntersections[i] + t*(mYvaluesOfIntersections[i + 1] - mYvaluesOfIntersections[i]);
				closestPointsXvalues[i] = closestPointXvalue;
				closestPointsYvalues[i] = closestPointYvalue;
				minDistances[i] = pow((x - closestPointXvalue), 2) + pow((y - closestPointYvalue), 2);
			}
			int indexOfMin = 0;
			for (i = 1; i < NumOfsegments; i++)
				indexOfMin = (minDistances[i] < minDistances[indexOfMin] ? i : indexOfMin);

			x = closestPointsXvalues[indexOfMin];
			y = closestPointsYvalues[indexOfMin];

			nu_f_cur.x = (x / prevX)*(nu_f_cur.x);
			nu_f_cur.y = (x / prevX)*(nu_f_cur.y);
			x_vec[index + numElements] = nu_f_cur;

			log_fz_cur.x = y;
			log_fz_cur.y = log_fz_cur.y;
			x_vec[index] = log_fz_cur;

		}//else - update if point is in polygon somehow
	}
}

#endif